#include "hip/hip_runtime.h"
#include "hello_world.h"
#include <stdio.h>

typedef struct firewallNode{
    uint8_t src_ip[4];
    // uint8_t dst_ip[4];
    uint8_t mask;
}firewallNode;

firewallNode *hst_states;
firewallNode *dev_states;
int count_firewallNodes;
__global__ void mac_swap_kernel(packet_hdrs *hst_hdrs, uint64_t size,firewallNode *dev_states, int states_count){
	int tid=blockDim.x * blockIdx.x + threadIdx.x;
    int a=1;
	if(tid<size){
        uint8_t tmp_dst[4];
        uint8_t shouldBlock = 0;
        for(int i = 0; i < states_count; i+=1) {
            memcpy(&tmp_dst, &hst_hdrs[tid].ipHdr.dst_ip, sizeof(uint8_t)*4);
            int mask = dev_states[i].mask;
            uint8_t mask_bits[4];
            for(int j = 0; j < mask/8; j+=1) {
                mask_bits[j] = 255;
            }
            if(mask%8 != 0) {
                int bit = mask/8;
                int mask_val = 0;
                for(int j = 0; j < mask%8; j+=1) {
                    mask_val+=(1<<(8-j));
                }
                mask_bits[bit] = mask_val;
            }

            for(int j = 0; j < 4; j+=1) {
                tmp_dst[j]&=mask_bits[j];
            }

            uint8_t matchesIp = 1;
            for(int j = 0; j < 4; j+=1) {
                if(tmp_dst[j] != dev_states[i].src_ip[j])
                    matchesIp = 0;
            }
            if(matchesIp) {
                shouldBlock = 1;
            }
        }
	}
}

void gpu_kernel_call(packet_hdrs *dev_hdrs,uint64_t size){
    int numblocks=(size/32)+1;
     printf("here\n");
    // printf("hehre::%llu,%d\n",size,numblocks);
    mac_swap_kernel<<<numblocks,32>>>(dev_hdrs, size,dev_states,count_firewallNodes);
     // printf("hehre::%llu\n",size);
}

void cpu_nf_call(packet_hdrs *pack_hdr)
{
    uint8_t tmp_dst[4];
    uint8_t shouldBlock = 0;
    for(int i = 0; i < count_firewallNodes; i+=1) {
        memcpy(&tmp_dst, pack_hdr->ipHdr.dst_ip, sizeof(uint8_t)*4);
        int mask = hst_states[i].mask;
        uint8_t mask_bits[4];
        for(int j = 0; j < mask/8; j+=1) {
            mask_bits[j] = 255;
        }
        if(mask%8 != 0) {
            int bit = mask/8;
            int mask_val = 0;
            for(int j = 0; j < mask%8; j+=1) {
                mask_val+=(1<<(8-j));
            }
            mask_bits[bit] = mask_val;
        }

        for(int j = 0; j < 4; j+=1) {
            tmp_dst[j]&=mask_bits[j];
        }

        uint8_t matchesIp = 1;
        for(int j = 0; j < 4; j+=1) {
            if(tmp_dst[j] != hst_states[i].src_ip[j])
                matchesIp = 0;
        }
        if(matchesIp) {
            shouldBlock = 1;
        }
    }
}

int init(firewallNode** hst_blockList)
{
    FILE *fptr;
    fptr = fopen ("blockList","r");
    fscanf(fptr,"%d\n",&count_firewallNodes) ;
    // printf("count found!! %d\n", count_firewallNodes);
    *hst_blockList=(firewallNode *)malloc(count_firewallNodes*sizeof(firewallNode));
    hst_states=*hst_blockList;
    for(int i=0;i<count_firewallNodes;i++)
    {
        fscanf(fptr,"%hhu.%hhu.%hhu.%hhu/%hhu\n",&hst_states[i].src_ip[0],&hst_states[i].src_ip[1],
            &hst_states[i].src_ip[2],&hst_states[i].src_ip[3],&hst_states[i].mask);
    }
    return count_firewallNodes;
}

void init_gpu_state(firewallNode* dev_blockList)
{
    dev_states=dev_blockList;

}
