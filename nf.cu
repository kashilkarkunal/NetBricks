#include "hip/hip_runtime.h"
#include "hello_world.h"

__global__ void mac_swap_kernel(packet_hdrs *hst_hdrs, uint64_t size){
	int tid=threadIdx.x;
	if(tid<size){
        uint8_t tmp[6];
        memcpy(&tmp,&hst_hdrs[tid].ethHdr.src_address,6);
        memcpy(&hst_hdrs[tid].ethHdr.src_address,&hst_hdrs[tid].ethHdr.dst_address,6);
        memcpy(&hst_hdrs[tid].ethHdr.dst_address,&tmp,6);
	}
}

void gpu_kernel_call(packet_hdrs *dev_hdrs,uint64_t size){
    mac_swap_kernel<<<1,size>>>(dev_hdrs, size);
}

void cpu_nf_call(packet_hdrs *pack_hdr)
{
    uint8_t tmp[6];
        // for(int j=0;j<6;j++)
        //     printf("%02x::",(*pck_hdrs).ethHdr.src_address[j] );
        // printf("\n");
    memcpy(&tmp,pack_hdr->ethHdr.src_address,6);
    memcpy(pack_hdr->ethHdr.src_address,pack_hdr->ethHdr.dst_address,6);
    memcpy(pack_hdr->ethHdr.dst_address,&tmp,6);
}
