#include "hip/hip_runtime.h"
#include "hello_world.h"
typedef struct firewallNode{
    uint8_t src_ip[4];
    uint8_t dst_ip[4];
    int mask[4];
}firewallNode;

firewallNode *blockIpsList;

__global__ void mac_swap_kernel(packet_hdrs *hst_hdrs, uint64_t size){
	int tid=threadIdx.x;
    int a=1;
	if(tid<size){
        uint8_t tmp[6];
        for(int k=0;k<100;k++)
            a++;
        memcpy(&tmp,&hst_hdrs[tid].ethHdr.src_address,6);
        memcpy(&hst_hdrs[tid].ethHdr.src_address,&hst_hdrs[tid].ethHdr.dst_address,6);
        memcpy(&hst_hdrs[tid].ethHdr.dst_address,&tmp,6);
	}
}

void gpu_kernel_call(packet_hdrs *dev_hdrs,uint64_t size){
    int numblocks=(size/32)+1;
    mac_swap_kernel<<<numblocks,32>>>(dev_hdrs, size);
}

void cpu_nf_call(packet_hdrs *pack_hdr)
{
    uint8_t tmp[6];
    int a=0;
    for(int k=0;k<100;k++)
            a++;
    memcpy(&tmp,pack_hdr->ethHdr.src_address,6);
    memcpy(pack_hdr->ethHdr.src_address,pack_hdr->ethHdr.dst_address,6);
    memcpy(pack_hdr->ethHdr.dst_address,&tmp,6);
}

void init()
{
    // File *fptr;
    // fptr = fopen ("blockList.txt","r");

 
}
