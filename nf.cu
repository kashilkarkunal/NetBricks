#include "hip/hip_runtime.h"
#include "hello_world.h"
#include <stdio.h>

typedef struct firewallNode{
    uint8_t src_ip[4];
    // uint8_t dst_ip[4];
    uint8_t mask;
}firewallNode;

firewallNode *hst_states;
firewallNode *dev_states;
int count_firewallNodes;
__global__ void mac_swap_kernel(packet_hdrs *hst_hdrs, uint64_t size,firewallNode *dev_states, int states_count){
	int tid=blockDim.x * blockIdx.x + threadIdx.x;
    int a=1;
    // printf("%d\n",tid );
    // if(tid==0)
    //     for(int i=0;i<states_count;i++)
    //         printf("%u.%u.%u.%u/%u\n", dev_states[i].src_ip[0],dev_states[i].src_ip[1],
    //             dev_states[i].src_ip[2],dev_states[i].src_ip[3],dev_states[i].mask);
	if(tid<size){

        // printf("gp::%d::%llu\n",tid,size );
        uint8_t tmp[6];
        
        for(int k=0;k<500;k++)
            a++;
        memcpy(&tmp,&hst_hdrs[tid].ethHdr.src_address,6);
        memcpy(&hst_hdrs[tid].ethHdr.src_address,&hst_hdrs[tid].ethHdr.dst_address,6);
        memcpy(&hst_hdrs[tid].ethHdr.dst_address,&tmp,6);
	}
}

void gpu_kernel_call(packet_hdrs *dev_hdrs,uint64_t size){
    int numblocks=(size/32)+1;
    // printf("here\n");
    // printf("hehre::%llu,%d\n",size,numblocks);
    mac_swap_kernel<<<numblocks,32>>>(dev_hdrs, size,dev_states,count_firewallNodes);
     // printf("hehre::%llu\n",size);
}

void cpu_nf_call(packet_hdrs *pack_hdr)
{
    uint8_t tmp[6];
    int a=0;
    for(int k=0;k<500;k++)
            a++;
    memcpy(&tmp,pack_hdr->ethHdr.src_address,6);
    memcpy(pack_hdr->ethHdr.src_address,pack_hdr->ethHdr.dst_address,6);
    memcpy(pack_hdr->ethHdr.dst_address,&tmp,6);
}

int init(firewallNode** hst_blockList)
{
    FILE *fptr;
    fptr = fopen ("blockList","r");
    fscanf(fptr,"%d\n",&count_firewallNodes) ;
    // printf("count found!! %d\n", count_firewallNodes);
    *hst_blockList=(firewallNode *)malloc(count_firewallNodes*sizeof(firewallNode));
    hst_states=*hst_blockList;
    for(int i=0;i<count_firewallNodes;i++)
    {
        fscanf(fptr,"%hhu.%hhu.%hhu.%hhu/%hhu\n",&hst_states[i].src_ip[0],&hst_states[i].src_ip[1],
            &hst_states[i].src_ip[2],&hst_states[i].src_ip[3],&hst_states[i].mask);
    }
    return count_firewallNodes;
}

void init_gpu_state(firewallNode* dev_blockList)
{
    dev_states=dev_blockList;

}
