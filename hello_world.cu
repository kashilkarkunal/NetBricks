#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
// #include "hello_world.h"
#include <arpa/inet.h>
#include<pthread.h>
#include "nf.cu"

#define timeIO 0
#define RunLevel 1

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))

int kunal=5678;
uint64_t Max_CPU_BatchSize=20;
uint64_t Cpu_BatchSize;
uint64_t GPU_BatchSize;
uint64_t max_size=512;
packet_hdrs *hst_hdrs;
packet_hdrs* dev_hdrs;
int first=0;
hipError_t err = hipSuccess; 
size_t size_dev_hdrs;

firewallNode *hst_state;
firewallNode *dev_state;


unsigned long long diff(unsigned lo1,unsigned hi1, unsigned lo2, unsigned hi2)
{
    unsigned long long int a=( (unsigned long long)lo1)|( ((unsigned long long)hi1)<<32 );
    unsigned long long int b=( (unsigned long long)lo2)|( ((unsigned long long)hi2)<<32 );
    return b-a;
}

typedef struct timer_struct{
    unsigned cpu_hi1,cpu_hi2,cpu_lo1,cpu_lo2;
    unsigned gpu_mem_hi1,gpu_mem_hi2,gpu_mem_lo1,gpu_mem_lo2;
    unsigned gpu_hi1,gpu_hi2,gpu_lo1,gpu_lo2;
} timer_struct;

timer_struct timers;

//the time measurement code was inspired from https://www.mcs.anl.gov/~kazutomo/rdtsc.html



typedef struct pthread_struct {
    GPUMbuf **packetStream;
    uint64_t size;
} pthread_struct;

packet_hdrs *packet_hdr_ptr(GPUMbuf **packetStream,int i){
    GPUMbuf *mbuf= packetStream[i];
    // printf("%d\n", (*mbuf).pkt_len);
    packet_hdrs *buf=(packet_hdrs *)((*mbuf).buf_addr+(*mbuf).data_off);
    return buf;
}



void *cpu_nf_caller_call(void *arg){
    pthread_struct *args=(pthread_struct *)arg;
    GPUMbuf **packetStream=args->packetStream;
    uint64_t size=args->size;
    for(int i=0;i<size;i++){
        packet_hdrs *pck_hdrs=packet_hdr_ptr(packetStream,i);
        cpu_nf_call(pck_hdrs);
    }
    return NULL;
}

void cpu_nf_caller_call_justCPU(GPUMbuf **packetStream, uint64_t size){
    // pthread_struct *args=(pthread_struct *)arg;
    // GPUMbuf **packetStream=args->packetStream;
    // uint64_t size=args->size;
    for(int i=0;i<size;i++){
        packet_hdrs *pck_hdrs=packet_hdr_ptr(packetStream,i);
        cpu_nf_call(pck_hdrs);
    }
    return;
}

void GPU_startTime()
{
    asm volatile (
      "rdtsc\n\t"
      "mov %%edx, %0\n\t"
      "mov %%eax, %1\n\t"
      : "=r" (timers.gpu_hi1), "=r" (timers.gpu_lo1)
      :: "%rax", "%rbx", "%rcx", "%rdx");

}

void GPU_endTime()
{
  asm volatile ("cpuid\n\t"
  "rdtsc\n\t"
  "mov %%edx, %0\n\t"
  "mov %%eax, %1\n\t"
  : "=r" (timers.gpu_hi2), "=r" (timers.gpu_lo2)
  :: "%rax", "%rbx", "%rcx", "%rdx");
}

void CPU_startTime()
{
  asm volatile (
      "rdtsc\n\t"
      "mov %%edx, %0\n\t"
      "mov %%eax, %1\n\t"
      : "=r" (timers.cpu_hi1), "=r" (timers.cpu_lo1)
      :: "%rax", "%rbx", "%rcx", "%rdx");
}

void CPU_endTime()
{
  asm volatile (
  "rdtsc\n\t"
  "mov %%edx, %0\n\t"
  "mov %%eax, %1\n\t"
  : "=r" (timers.cpu_hi2), "=r" (timers.cpu_lo2)
  :: "%rax", "%rbx", "%rcx", "%rdx");
}

void GPU_Tot_startTime()
{
  asm volatile (
        "rdtsc\n\t"
        "mov %%edx, %0\n\t"
        "mov %%eax, %1\n\t"
        : "=r" (timers.gpu_mem_hi1), "=r" (timers.gpu_mem_lo1)
        :: "%rax", "%rbx", "%rcx", "%rdx");

}

void GPU_Tot_endTime()
{
        asm volatile (
        "rdtsc\n\t"
        "mov %%edx, %0\n\t"
        "mov %%eax, %1\n\t"
        : "=r" (timers.gpu_mem_hi2), "=r" (timers.gpu_mem_lo2)
        :: "%rax", "%rbx", "%rcx", "%rdx");

}
extern "C" {
void swap_mac_address(GPUMbuf **packetStream, uint64_t size){
  if(size==0)
    return;

  if(!first){
    size_dev_hdrs=max_size*sizeof(packet_hdrs);
    hst_hdrs=(packet_hdrs *)malloc(max_size*sizeof(packet_hdrs));
    int count_states=init(&hst_state);
    if(RunLevel>1){
      GPU_startTime();
      err=hipMalloc((void **)&dev_hdrs, size_dev_hdrs);
      if (err != hipSuccess){
          fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }
      size_t size_dev_state=count_states*sizeof(firewallNode);

      //for(int i=0;i<count_states;i++)
           //printf("%hhu.%hhu.%hhu.%hhu/%hhu\n", hst_state[i].src_ip[0],hst_state[i].src_ip[1],
        //hst_state[i].src_ip[2],hst_state[i].src_ip[3],hst_state[i].mask);



      //printf("count::%d\n",count_states );
      err=hipMalloc((void **)&dev_state, size_dev_state);
      if (err != hipSuccess){
          fprintf(stderr, "Failed to allocate device vector for states(error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }
      err = hipMemcpy(dev_state, hst_state,size_dev_state, hipMemcpyHostToDevice);
      if (err != hipSuccess)
      {
          fprintf(stderr, "Failed to copy states to GPU device (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }
      init_gpu_state(dev_state);
      GPU_endTime();  
      if(timeIO)
        printf("CUDA MALLOC::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));
    } 
    
    first=1; 
  }
  unsigned long long int cpu_time=0;
  unsigned long long int gpu_time=0;
  unsigned long long int hyb_time=0;

  if(RunLevel==2)
    Cpu_BatchSize=0;
  else
    Cpu_BatchSize=MIN(Max_CPU_BatchSize,size);


  // if(timeIO)
  // {
  //   // printf("TOT_BS::%llu\n",size );
  //   // printf("CPU_BS::%llu\n",Cpu_BatchSize );
  // }
  

  if(RunLevel==1){
    CPU_startTime();

    // pthread_t my_thread;
    // pthread_struct pthread_Args;
    // pthread_Args.packetStream=packetStream;
    // pthread_Args.size=size;
    // pthread_create(&my_thread, NULL, cpu_nf_caller_call, &pthread_Args); 
    // pthread_join(my_thread, NULL);
    cpu_nf_caller_call_justCPU(packetStream,size);

    CPU_endTime();
    cpu_time=diff(timers.cpu_lo1,timers.cpu_hi1,timers.cpu_lo2,timers.cpu_hi2);
  }
  else if(RunLevel==3){
    CPU_startTime();
    pthread_t my_thread2;
    pthread_struct pthread_Args2;
    pthread_Args2.packetStream=packetStream;
    pthread_Args2.size=Cpu_BatchSize;
   
    pthread_create(&my_thread2, NULL, cpu_nf_caller_call, &pthread_Args2); 
    

    if(size>Cpu_BatchSize)
      GPU_BatchSize=size-Cpu_BatchSize;
    else
      GPU_BatchSize=0;

    // if(timeIO)
      // printf("GPU_BS::%llu\n",GPU_BatchSize );

    GPU_Tot_startTime();
    GPU_startTime();
    for(int i=0;i<GPU_BatchSize;i++){
        GPUMbuf mbuf=*(packetStream[i+Cpu_BatchSize]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
    }
    size_t GPUSize=GPU_BatchSize*sizeof(packet_hdrs);
    GPU_endTime();
    if(timeIO)
      printf("CPU Copy time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));
      
    GPU_startTime();
    err = hipMemcpy(dev_hdrs, hst_hdrs,GPUSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy to GPU device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    GPU_endTime();

    if(timeIO)
      printf("GPU Copy time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));

    GPU_startTime();
    

    gpu_kernel_call(dev_hdrs, GPU_BatchSize);  
    err=hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch mac_swap_kernel kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    GPU_endTime();
    if(timeIO)
      printf("GPU Run time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));
    
    GPU_startTime();
    err = hipMemcpy(hst_hdrs, dev_hdrs, GPUSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
         fprintf(stderr, "Failed to copy vectorC from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
      // hipFree(dev_hdrs);
    GPU_endTime();
    if(timeIO)
      printf("GPU copy back time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));

    GPU_startTime();
    for(int i=0;i<GPU_BatchSize;i++){
        GPUMbuf mbuf=*(packetStream[i+Cpu_BatchSize]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        // memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
        memcpy(buf,(uint8_t*)&hst_hdrs[i],sizeof(packet_hdrs));
    }
    GPU_endTime();
    if(timeIO)
      printf("CPU Copy back time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));
    GPU_Tot_endTime();
    pthread_join(my_thread2, NULL);
    CPU_endTime();
    hyb_time=diff(timers.cpu_lo1,timers.cpu_hi1,timers.cpu_lo2,timers.cpu_hi2);

    gpu_time=diff(timers.gpu_mem_lo1,timers.gpu_mem_hi1,timers.gpu_mem_lo2,timers.gpu_mem_hi2);
  }
  else{
    if(size>Cpu_BatchSize)
      GPU_BatchSize=size-Cpu_BatchSize;
    else
      GPU_BatchSize=0;

    // printf("GPU_BS::%llu\n",GPU_BatchSize );

    GPU_Tot_startTime();
    GPU_startTime();
    for(int i=0;i<GPU_BatchSize;i++){
        GPUMbuf mbuf=*(packetStream[i+Cpu_BatchSize]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
    }
    size_t GPUSize=GPU_BatchSize*sizeof(packet_hdrs);
    GPU_endTime();
    if(timeIO)
      printf("CPU Copy time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));
      
    GPU_startTime();
    err = hipMemcpy(dev_hdrs, hst_hdrs,GPUSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy to GPU device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    GPU_endTime();

    if(timeIO)
      printf("GPU Copy time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));

    GPU_startTime();
    

    gpu_kernel_call(dev_hdrs, GPU_BatchSize);  
    err=hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch mac_swap_kernel kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    GPU_endTime();
    if(timeIO)
      printf("GPU Run time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));
    
    GPU_startTime();
    err = hipMemcpy(hst_hdrs, dev_hdrs, GPUSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
         fprintf(stderr, "Failed to copy vectorC from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
      // hipFree(dev_hdrs);
    GPU_endTime();
    if(timeIO)
      printf("GPU copy back time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));

    GPU_startTime();
    for(int i=0;i<GPU_BatchSize;i++){
        GPUMbuf mbuf=*(packetStream[i+Cpu_BatchSize]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        // memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
        memcpy(buf,(uint8_t*)&hst_hdrs[i],sizeof(packet_hdrs));
    }
    GPU_endTime();
    if(timeIO)
      printf("CPU Copy back time::%llu\n", diff(timers.gpu_lo1,timers.gpu_hi1,timers.gpu_lo2,timers.gpu_hi2));
    GPU_Tot_endTime();
    CPU_endTime();
    hyb_time=diff(timers.cpu_lo1,timers.cpu_hi1,timers.cpu_lo2,timers.cpu_hi2);

    gpu_time=diff(timers.gpu_mem_lo1,timers.gpu_mem_hi1,timers.gpu_mem_lo2,timers.gpu_mem_hi2);
  }
  if(timeIO){
    printf("GPU::%llu\n", gpu_time);
    printf("CPU::%llu\n",cpu_time );
    printf("HYB::%llu,\n",hyb_time);
  }
}


}

