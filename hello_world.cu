#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

typedef struct _packet_{
    char src_address[6];
    char dst_address[6];
    char data[300];
} packet;


packet create_packet() {
    packet p;
    for(int i = 0; i < 5; i+=1) {
        char srch = (rand()%26) + 65;
        char dstch = (rand()%26) + 65;

        p.src_address[i] = srch;
        p.dst_address[i] = dstch;
    }

    p.src_address[5] = '\0';
    p.dst_address[5] = '\0';

    for(int i = 0; i < 100; i+=1) {
        char data = (rand()%26) + 65;
        p.data[i] = data;
    }

    return p;
}

__global__ void VecAdd(packet *A, int n) {
    
    int tx = threadIdx.x, ty = threadIdx.y;

    int I = blockIdx.y*blockDim.y + ty;
    int J = blockIdx.x*blockDim.x + tx;

    int i = I*n + J;

    if( i < n) { 
        for(int j = 0; j < 5; j+=1) {
            A[i].src_address[j] = 'a';
            A[i].dst_address[j] = 'b';
        }
    }
}

extern "C" {
void garble_packet() {

    packet packets[100];
    for(int i = 0; i < 100; i+=1) {
        packets[i] = create_packet();
    }

    int size = 100*sizeof(packet);

    packet *a;
    hipMalloc((void **)&a, size);

    hipMemcpy(a, packets, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    VecAdd<<<10, 10>>> (a, 100);

    hipDeviceSynchronize();
    hipMemcpy(packets, a, size, hipMemcpyDeviceToHost);
    hipFree(a);

    for(int i = 0; i < 100; i+=1) 
        printf("%s : %s \n", packets[i].src_address, packets[i].dst_address);
}
}
