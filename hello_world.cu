#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include "hello_world.h"

packet create_packet() {
    packet p;
    for(int i = 0; i < 5; i+=1) {
        char srch = (rand()%26) + 65;
        char dstch = (rand()%26) + 65;

        p.src_address[i] = srch;
        p.dst_address[i] = dstch;
    }

    p.src_address[5] = '\0';
    p.dst_address[5] = '\0';

    for(int i = 0; i < 100; i+=1) {
        char data = (rand()%26) + 65;
        p.data[i] = data;
    }

    return p;
}

__global__ void VecAdd(packet *A, int n) {
    
    int tx = threadIdx.x, ty = threadIdx.y;

    int I = blockIdx.y*blockDim.y + ty;
    int J = blockIdx.x*blockDim.x + tx;

    int i = I*n + J;

    if( i < n) { 
        for(int j = 0; j < 5; j+=1) {
            A[i].src_address[j] = 'a';
            A[i].dst_address[j] = 'b';
        }
    }
}

__global__ void mac_swap_kernel(GPUMbuf *packetStream, uint64_t size){
	int tid=threadIdx.x;
	if(tid<size){
		printf("GPU %d %lld %lld %lld \n", tid, packetStream[tid].pkt_len,  packetStream[tid].buf_addr,  packetStream[tid].phys_addr,  packetStream[tid].data_off);

	}
	//todo::actual macswap???
}

extern "C" {
void swap_mac_address(GPUMbuf **packetStream, uint64_t size){
    hipError_t err = hipSuccess;
    hipDeviceReset();

 	err = hipSetDeviceFlags(hipDeviceMapHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to set flag %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    err = hipDeviceSynchronize();
	if (err != hipSuccess){
		fprintf(stderr, "Failed to set flag %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    GPUMbuf *dev_stream;
    GPUMbuf *stream;
   	err = hipHostMalloc((void**)&stream, size*sizeof(GPUMbuf)); 
	if (err != hipSuccess){
		fprintf(stderr, "Failed cuda hipHostAlloc(error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipHostGetDevicePointer( &dev_stream, stream, 0 );
	if (err != hipSuccess){
		fprintf(stderr, "Failed cuda hipHostGetDevicePointer(error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err=hipDeviceSynchronize();
	for (int i=0; i<size; i++) {
        stream[i]=*(packetStream[i]);
        printf("Outside GPU Size %d\n", stream[i].pkt_len);
        print_data(stream[i].buf_addr, stream[i].pkt_len);
    }

	mac_swap_kernel<<<1,size>>>(dev_stream, size);

    err = hipGetLastError();
	if (err != hipSuccess){
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    err=hipDeviceSynchronize();
	if (err != hipSuccess){
		fprintf(stderr, "waitinf for cuda kernel fialed (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	for(int i=0;i<size;i++)
		*packetStream[i]=stream[i];

	hipHostFree( stream );
}

void print_data(uint8_t* buf_addr, uint32_t pkt_len){

        int i = 0;
        for( ; i < pkt_len; ++i )
            printf("inbuf %lld ", buf_addr[i]);

}

void garble_packet(packet packets[], int num) {


    printf("Starting Cuda Program %d \n", num);
    int size = num*sizeof(packet);

    packet *a;
    hipMalloc((void **)&a, size);

    hipMemcpy(a, packets, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    VecAdd<<<num/10, 10>>> (a, num);

    hipDeviceSynchronize();
    hipMemcpy(packets, a, size, hipMemcpyDeviceToHost);
    hipFree(a);

    for(int i = 0; i < num; i+=1) 
        printf("%s : %s \n", packets[i].src_address, packets[i].dst_address);
}



/*
void swap_mac_address(GPUMbuf **packets, uint64_t size) {
    if(packets == NULL) {
        printf("Packets are null");
        return;
    }
    for(int i = 0; i < size; i+=1) {
        GPUMbuf *packet = packets[i];
        if(packet == NULL) {
            printf("Packet is null inside loop\n");
            continue;
        }

        uint8_t *buf_addr = (*packet).buf_addr;
        printf("The buff addr is %s\n", (char *)buf_addr);
        printf("The pkt_len size is %d\n", (*packet).pkt_len);
        printf("The buf_len size is %d\n", (*packet).buf_len);
        printf("The timestamp size is %lld\n", (*packet).timestamp);

        for(int i = 0; i < (*packet).pkt_len; i+=1) {
            printf("%c", (char) packet++);
        }
    }
}
*/
}

