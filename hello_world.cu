#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include "hello_world.h"

packet create_packet() {
    packet p;
    for(int i = 0; i < 5; i+=1) {
        char srch = (rand()%26) + 65;
        char dstch = (rand()%26) + 65;

        p.src_address[i] = srch;
        p.dst_address[i] = dstch;
    }

    p.src_address[5] = '\0';
    p.dst_address[5] = '\0';

    for(int i = 0; i < 100; i+=1) {
        char data = (rand()%26) + 65;
        p.data[i] = data;
    }

    return p;
}

__global__ void VecAdd(packet *A, int n) {
    
    int tx = threadIdx.x, ty = threadIdx.y;

    int I = blockIdx.y*blockDim.y + ty;
    int J = blockIdx.x*blockDim.x + tx;

    int i = I*n + J;

    if( i < n) { 
        for(int j = 0; j < 5; j+=1) {
            A[i].src_address[j] = 'a';
            A[i].dst_address[j] = 'b';
        }
    }
}

extern "C" {
void garble_packet(packet packets[], int num) {


    printf("Starting Cuda Program %d \n", num);
    int size = num*sizeof(packet);

    packet *a;
    hipMalloc((void **)&a, size);

    hipMemcpy(a, packets, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    VecAdd<<<num/10, 10>>> (a, num);

    hipDeviceSynchronize();
    hipMemcpy(packets, a, size, hipMemcpyDeviceToHost);
    hipFree(a);

    for(int i = 0; i < num; i+=1) 
        printf("%s : %s \n", packets[i].src_address, packets[i].dst_address);
}

void swap_mac_address(GPUMbuf **packets, uint64_t size) {
    if(packets == NULL) {
        printf("Packets are null");
        return;
    }
    for(int i = 0; i < size; i+=1) {
        GPUMbuf *packet = packets[i];
        if(packet == NULL) {
            printf("Packet is null inside loop\n");
            continue;
        }
        uint8_t *buf_addr = (*packet).buf_addr;
        printf("The pool size is %s\n", buf_addr);
        printf("The pkt_len size is %d\n", (*packet).pkt_len);
        printf("The buf_len size is %d\n", (*packet).buf_len);
        printf("The pkt_len size is %lld\n", (*packet).timestamp);
    }
}
}

