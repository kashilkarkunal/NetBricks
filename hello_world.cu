#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include "hello_world.h"
#include <arpa/inet.h>


packet create_packet() {
    packet p;
    for(int i = 0; i < 5; i+=1) {
        char srch = (rand()%26) + 65;
        char dstch = (rand()%26) + 65;

        p.src_address[i] = srch;
        p.dst_address[i] = dstch;
    }

    p.src_address[5] = '\0';
    p.dst_address[5] = '\0';

    for(int i = 0; i < 100; i+=1) {
        char data = (rand()%26) + 65;
        p.data[i] = data;
    }

    return p;
}

__global__ void VecAdd(packet *A, int n) {
    
    int tx = threadIdx.x, ty = threadIdx.y;

    int I = blockIdx.y*blockDim.y + ty;
    int J = blockIdx.x*blockDim.x + tx;

    int i = I*n + J;

    if( i < n) { 
        for(int j = 0; j < 5; j+=1) {
            A[i].src_address[j] = 'a';
            A[i].dst_address[j] = 'b';
        }
    }
}

__global__ void mac_swap_kernel(packet_hdrs *hst_hdrs, uint64_t size){
	int tid=threadIdx.x;
	if(tid<size)
    {
  //       printf("GPU::");
		// // printf("GPU DATA %d %lld %lld %lld \n", tid, packetStream[tid].pkt_len,  packetStream[tid].buf_addr,  packetStream[tid].phys_addr,  packetStream[tid].data_off);
  //       for(int j=0;j<6;j++)
  //           printf("%02x::", hst_hdrs[tid].ethHdr.dst_address[j]);
  //       printf("<---->");
  //       for(int j=0;j<6;j++)
  //           printf("%02x::", hst_hdrs[tid].ethHdr.src_address[j]);
  //       printf("\n");
        for(int i=0;i<6;i++)
        {
            uint8_t tmp=hst_hdrs[tid].ethHdr.src_address[i];
            hst_hdrs[tid].ethHdr.src_address[i]=hst_hdrs[tid].ethHdr.dst_address[i];
            hst_hdrs[tid].ethHdr.dst_address[i]=tmp;
        }
	}
	//todo::actual macswap???
}

extern "C" {
void swap_mac_address(GPUMbuf **packetStream, uint64_t size){
    hipError_t err = hipSuccess;
    hipDeviceReset();
    packet_hdrs hst_hdrs[size];
    packet_hdrs* dev_hdrs;
    for(int i=0;i<size;i++)
    {
        GPUMbuf mbuf=*(packetStream[i]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
        // for(int j=0;j<6;j++)
        //     printf("%02x::", hst_hdrs[i].ethHdr.dst_address[j]);
        // printf("<---->");
        // for(int j=0;j<6;j++)
        //     printf("%02x::", hst_hdrs[i].ethHdr.src_address[j]);
        // printf("\n");
        // struct in_addr ip_addr;
        // ip_addr.s_addr=;
        //  memcpy(&ip_addr.s_addr, &hst_hdrs[i].ipHdr.src_ip, 4);
        // printf("\nThe src IP address is %s\n", inet_ntoa(ip_addr));
        // memcpy(&ip_addr.s_addr, &hst_hdrs[i].ipHdr.dst_ip, 4);
        // printf("\nThe src IP address is %s\n", inet_ntoa(ip_addr));
    }
    size_t size_dev_hdrs=size*sizeof(packet_hdrs);
    err = hipMalloc((void **)&dev_hdrs, size_dev_hdrs);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dev_hdrs, hst_hdrs,size_dev_hdrs, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy to GPU device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipDeviceSynchronize();

    mac_swap_kernel<<<1,size>>>(dev_hdrs, size);
    
    err=hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch mac_swap_kernel kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(hst_hdrs, dev_hdrs, size_dev_hdrs, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
         fprintf(stderr, "Failed to copy vectorC from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

     for(int i=0;i<size;i++)
    {
        GPUMbuf mbuf=*(packetStream[i]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        // memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
        memcpy(buf,(uint8_t*)&hst_hdrs[i],sizeof(packet_hdrs));
    }

 // 	err = hipSetDeviceFlags(hipDeviceMapHost);
	// if (err != hipSuccess){
	// 	fprintf(stderr, "Failed to set flag %s)!\n", hipGetErrorString(err));
	// 	exit(EXIT_FAILURE);
	// }

 //    err = hipDeviceSynchronize();
	// if (err != hipSuccess){
	// 	fprintf(stderr, "Failed to set flag %s)!\n", hipGetErrorString(err));
	// 	exit(EXIT_FAILURE);
	// }

    // GPUMbuf *dev_stream;
    
	// if (err != hipSuccess){
	// 	fprintf(stderr, "Failed cuda hipHostAlloc(error code %s)!\n", hipGetErrorString(err));
	// 	exit(EXIT_FAILURE);
	// }
	// err = hipHostGetDevicePointer( &dev_stream, stream, 0 );
	// if (err != hipSuccess){
	// 	fprintf(stderr, "Failed cuda hipHostGetDevicePointer(error code %s)!\n", hipGetErrorString(err));
	// 	exit(EXIT_FAILURE);
	// }
	// err=hipDeviceSynchronize();
    
    /*
    GPUMbuf *stream;
    hipHostMalloc((void**)&stream, size*sizeof(GPUMbuf)); 
	for (int i=0; i<size; i++) {
        stream[i]=*(packetStream[i]);
        printf("Outside GPU Size %d::%d::%d\n", stream[i].pkt_len,stream[i].data_len, stream[i].sync);
        int buff_dat = stream[i].data_off;
        for( ; buff_dat < stream[i].data_off+6; ++buff_dat )
            printf("%2x::", stream[i].buf_addr[buff_dat]);
        printf("<------->");
        for( ; buff_dat < stream[i].data_off+12; ++buff_dat )
            printf("%02x::", stream[i].buf_addr[buff_dat]);
        printf("==========");
        for( ; buff_dat < stream[i].data_off+14; ++buff_dat )
            printf("%02x::", stream[i].buf_addr[buff_dat]);
        struct in_addr ip_addr;
        // ip_addr.s_addr = *((int*)(stream[i].buf_addr+buff_dat+3*4));
        memcpy(&ip_addr.s_addr, stream[i].buf_addr+buff_dat+3*4, 4);
        printf("\nThe src IP address is %s\n", inet_ntoa(ip_addr));
        memcpy(&ip_addr.s_addr, stream[i].buf_addr+buff_dat+4*4, 4);
        printf("The dst IP address is %s\n", inet_ntoa(ip_addr));

        printf("----%d::", stream[i].buf_addr[buff_dat+1]);
        printf("\n");
    }
    

	mac_swap_kernel<<<1,size>>>(dev_stream, size);

    err = hipGetLastError();
	if (err != hipSuccess){
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    err=hipDeviceSynchronize();
	if (err != hipSuccess){
		fprintf(stderr, "waitinf for cuda kernel fialed (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	for(int i=0;i<size;i++)
		*packetStream[i]=stream[i];

	hipHostFree( stream );
    */
}


void garble_packet(packet packets[], int num) {


    printf("Starting Cuda Program %d \n", num);
    int size = num*sizeof(packet);

    packet *a;
    hipMalloc((void **)&a, size);

    hipMemcpy(a, packets, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    VecAdd<<<num/10, 10>>> (a, num);

    hipDeviceSynchronize();
    hipMemcpy(packets, a, size, hipMemcpyDeviceToHost);
    hipFree(a);

    for(int i = 0; i < num; i+=1) 
        printf("%s : %s \n", packets[i].src_address, packets[i].dst_address);
}



/*
void swap_mac_address(GPUMbuf **packets, uint64_t size) {
    if(packets == NULL) {
        printf("Packets are null");
        return;
    }
    for(int i = 0; i < size; i+=1) {
        GPUMbuf *packet = packets[i];
        if(packet == NULL) {
            printf("Packet is null inside loop\n");
            continue;
        }

        uint8_t *buf_addr = (*packet).buf_addr;
        printf("The buff addr is %s\n", (char *)buf_addr);
        printf("The pkt_len size is %d\n", (*packet).pkt_len);
        printf("The buf_len size is %d\n", (*packet).buf_len);
        printf("The timestamp size is %lld\n", (*packet).timestamp);

        for(int i = 0; i < (*packet).pkt_len; i+=1) {
            printf("%c", (char) packet++);
        }
    }
}
*/
}

