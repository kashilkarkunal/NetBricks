#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
// #include "hello_world.h"
#include <arpa/inet.h>
#include<pthread.h>
#include "nf.cu"


packet_hdrs *packet_hdr_ptr(GPUMbuf **packetStream,int i){
    GPUMbuf *mbuf= packetStream[i];
    // printf("%d\n", (*mbuf).pkt_len);
    packet_hdrs *buf=(packet_hdrs *)((*mbuf).buf_addr+(*mbuf).data_off);
    return buf;
}



void cpu_nf_caller_call(GPUMbuf **packetStream,uint64_t size){
    for(int i=0;i<size;i++){
        packet_hdrs *pck_hdrs=packet_hdr_ptr(packetStream,i);
        cpu_nf_call(pck_hdrs);
        // uint8_t tmp[6];
        // // for(int j=0;j<6;j++)
        // //     printf("%02x::",(*pck_hdrs).ethHdr.src_address[j] );
        // // printf("\n");
        // memcpy(&tmp,pck_hdrs->ethHdr.src_address,6);
        // memcpy(pck_hdrs->ethHdr.src_address,pck_hdrs->ethHdr.dst_address,6);
        // memcpy(pck_hdrs->ethHdr.dst_address,&tmp,6);

    }
}




extern "C" {
void swap_mac_address(GPUMbuf **packetStream, uint64_t size){
    hipError_t err = hipSuccess;
    hipDeviceReset();
    packet_hdrs hst_hdrs[size];
    packet_hdrs* dev_hdrs;
    cpu_nf_caller_call(packetStream,size);
    for(int i=0;i<size;i++)
    {
        GPUMbuf mbuf=*(packetStream[i]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
        // for(int j=0;j<6;j++)
        //     printf("%02x::", hst_hdrs[i].ethHdr.dst_address[j]);
        // printf("<---->");
        // for(int j=0;j<6;j++)
        //     printf("%02x::", hst_hdrs[i].ethHdr.src_address[j]);
        // printf("\n");
        // struct in_addr ip_addr;
        // ip_addr.s_addr=;
        //  memcpy(&ip_addr.s_addr, &hst_hdrs[i].ipHdr.src_ip, 4);
        // printf("\nThe src IP address is %s\n", inet_ntoa(ip_addr));
        // memcpy(&ip_addr.s_addr, &hst_hdrs[i].ipHdr.dst_ip, 4);
        // printf("\nThe src IP address is %s\n", inet_ntoa(ip_addr));
    }
    size_t size_dev_hdrs=size*sizeof(packet_hdrs);
    err = hipMalloc((void **)&dev_hdrs, size_dev_hdrs);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dev_hdrs, hst_hdrs,size_dev_hdrs, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy to GPU device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipDeviceSynchronize();

    // gpu_kernel_call(dev_hdrs, size);
    
    
    err=hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch mac_swap_kernel kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(hst_hdrs, dev_hdrs, size_dev_hdrs, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
         fprintf(stderr, "Failed to copy vectorC from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

     for(int i=0;i<size;i++)
    {
        GPUMbuf mbuf=*(packetStream[i]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        // memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
        memcpy(buf,(uint8_t*)&hst_hdrs[i],sizeof(packet_hdrs));
    }

 // 	err = hipSetDeviceFlags(hipDeviceMapHost);
	// if (err != hipSuccess){
	// 	fprintf(stderr, "Failed to set flag %s)!\n", hipGetErrorString(err));
	// 	exit(EXIT_FAILURE);
	// }

 //    err = hipDeviceSynchronize();
	// if (err != hipSuccess){
	// 	fprintf(stderr, "Failed to set flag %s)!\n", hipGetErrorString(err));
	// 	exit(EXIT_FAILURE);
	// }

    // GPUMbuf *dev_stream;
    
	// if (err != hipSuccess){
	// 	fprintf(stderr, "Failed cuda hipHostAlloc(error code %s)!\n", hipGetErrorString(err));
	// 	exit(EXIT_FAILURE);
	// }
	// err = hipHostGetDevicePointer( &dev_stream, stream, 0 );
	// if (err != hipSuccess){
	// 	fprintf(stderr, "Failed cuda hipHostGetDevicePointer(error code %s)!\n", hipGetErrorString(err));
	// 	exit(EXIT_FAILURE);
	// }
	// err=hipDeviceSynchronize();
    
    /*
    GPUMbuf *stream;
    hipHostMalloc((void**)&stream, size*sizeof(GPUMbuf)); 
	for (int i=0; i<size; i++) {
        stream[i]=*(packetStream[i]);
        printf("Outside GPU Size %d::%d::%d\n", stream[i].pkt_len,stream[i].data_len, stream[i].sync);
        int buff_dat = stream[i].data_off;
        for( ; buff_dat < stream[i].data_off+6; ++buff_dat )
            printf("%2x::", stream[i].buf_addr[buff_dat]);
        printf("<------->");
        for( ; buff_dat < stream[i].data_off+12; ++buff_dat )
            printf("%02x::", stream[i].buf_addr[buff_dat]);
        printf("==========");
        for( ; buff_dat < stream[i].data_off+14; ++buff_dat )
            printf("%02x::", stream[i].buf_addr[buff_dat]);
        struct in_addr ip_addr;
        // ip_addr.s_addr = *((int*)(stream[i].buf_addr+buff_dat+3*4));
        memcpy(&ip_addr.s_addr, stream[i].buf_addr+buff_dat+3*4, 4);
        printf("\nThe src IP address is %s\n", inet_ntoa(ip_addr));
        memcpy(&ip_addr.s_addr, stream[i].buf_addr+buff_dat+4*4, 4);
        printf("The dst IP address is %s\n", inet_ntoa(ip_addr));

        printf("----%d::", stream[i].buf_addr[buff_dat+1]);
        printf("\n");
    }
    

	mac_swap_kernel<<<1,size>>>(dev_stream, size);

    err = hipGetLastError();
	if (err != hipSuccess){
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    err=hipDeviceSynchronize();
	if (err != hipSuccess){
		fprintf(stderr, "waitinf for cuda kernel fialed (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	for(int i=0;i<size;i++)
		*packetStream[i]=stream[i];

	hipHostFree( stream );
    */
}

}

