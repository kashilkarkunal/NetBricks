#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

typedef struct _packet_{
    char src_address[6];
    char dst_address[6];
    char data[6];
} packet;

packet create_packet() {
    packet p;
    for(int i = 0; i < 5; i+=1) {
        char srch = (rand()%26) + 65;
        char dstch = (rand()%26) + 65;

        p.src_address[i] = srch;
        p.dst_address[i] = dstch;
    }

    p.src_address[5] = '\0';
    p.dst_address[5] = '\0';

    for(int i = 0; i < 100; i+=1) {
        char data = (rand()%26) + 65;
        p.data[i] = data;
    }

    return p;
}

__global__ void VecAdd(packet *A, int n) {
    
    int tx = threadIdx.x, ty = threadIdx.y;

    int I = blockIdx.y*blockDim.y + ty;
    int J = blockIdx.x*blockDim.x + tx;

    int i = I*n + J;

    if( i < n) { 
        for(int j = 0; j < 5; j+=1) {
            A[i].src_address[j] = 'a';
            A[i].dst_address[j] = 'b';
        }
    }
}

extern "C" {
void garble_packet(packet packets[], int num) {


    printf("Starting Cuda Program %d \n", num);
    int size = num*sizeof(packet);

    packet *a;
    hipMalloc((void **)&a, size);

    hipMemcpy(a, packets, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    VecAdd<<<num/10, 10>>> (a, num);

    hipDeviceSynchronize();
    hipMemcpy(packets, a, size, hipMemcpyDeviceToHost);
    hipFree(a);

    for(int i = 0; i < num; i+=1) 
        printf("%s : %s \n", packets[i].src_address, packets[i].dst_address);
}
}

