#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
// #include "hello_world.h"
#include <arpa/inet.h>
#include<pthread.h>
#include "nf.cu"



typedef struct pthread_struct {
    GPUMbuf **packetStream;
    uint64_t size;
} pthread_struct;

packet_hdrs *packet_hdr_ptr(GPUMbuf **packetStream,int i){
    GPUMbuf *mbuf= packetStream[i];
    // printf("%d\n", (*mbuf).pkt_len);
    packet_hdrs *buf=(packet_hdrs *)((*mbuf).buf_addr+(*mbuf).data_off);
    return buf;
}



void *cpu_nf_caller_call(void *arg){
    pthread_struct *args=(pthread_struct *)arg;
    GPUMbuf **packetStream=args->packetStream;
    uint64_t size=args->size;
    for(int i=0;i<size;i++){
        packet_hdrs *pck_hdrs=packet_hdr_ptr(packetStream,i);
        cpu_nf_call(pck_hdrs);
    }
    return NULL;
}




extern "C" {
void swap_mac_address(GPUMbuf **packetStream, uint64_t size){
    hipError_t err = hipSuccess;
    hipDeviceReset();
    packet_hdrs hst_hdrs[size];
    packet_hdrs* dev_hdrs;

    pthread_t my_thread;
    pthread_struct pthread_Args;
    pthread_Args.packetStream=packetStream;
    pthread_Args.size=size;

    pthread_create(&my_thread, NULL, cpu_nf_caller_call, &pthread_Args); 
    pthread_join(my_thread, NULL);
    // cpu_nf_caller_call(packetStream,size);
    for(int i=0;i<size;i++)
    {
        GPUMbuf mbuf=*(packetStream[i]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
        // for(int j=0;j<6;j++)
        //     printf("%02x::", hst_hdrs[i].ethHdr.dst_address[j]);
        // printf("<---->");
        // for(int j=0;j<6;j++)
        //     printf("%02x::", hst_hdrs[i].ethHdr.src_address[j]);
        // printf("\n");
        // struct in_addr ip_addr;
        // ip_addr.s_addr=;
        //  memcpy(&ip_addr.s_addr, &hst_hdrs[i].ipHdr.src_ip, 4);
        // printf("\nThe src IP address is %s\n", inet_ntoa(ip_addr));
        // memcpy(&ip_addr.s_addr, &hst_hdrs[i].ipHdr.dst_ip, 4);
        // printf("\nThe src IP address is %s\n", inet_ntoa(ip_addr));
    }
    size_t size_dev_hdrs=size*sizeof(packet_hdrs);
    err = hipMalloc((void **)&dev_hdrs, size_dev_hdrs);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dev_hdrs, hst_hdrs,size_dev_hdrs, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy to GPU device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipDeviceSynchronize();

    // gpu_kernel_call(dev_hdrs, size);
    
    
    err=hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch mac_swap_kernel kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(hst_hdrs, dev_hdrs, size_dev_hdrs, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
         fprintf(stderr, "Failed to copy vectorC from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

     for(int i=0;i<size;i++)
    {
        GPUMbuf mbuf=*(packetStream[i]);
        uint8_t* buf=mbuf.buf_addr+mbuf.data_off;
        // memcpy((uint8_t*)&hst_hdrs[i],buf,sizeof(packet_hdrs));
        memcpy(buf,(uint8_t*)&hst_hdrs[i],sizeof(packet_hdrs));
    }
}

}

